
#include <hip/hip_runtime.h>
/// Multuply a matrix in CSR format with a dense vector. The vector is on the right hand side of the matrix.
/// @param[in] rows The number of rows of the matrix
/// @param[in] rowStart Array with length the number of rows + 1,
/// holding where each row starts in columnIndex and values arrays
/// @param[in] columnIndex elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]]
/// are the columns of the elements of the i-th row
/// @param[in] values elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]] are 
/// the values of the elements in the row.
/// @param[in] mult The vector which multiples the matrix (should not overlap with res)
/// @param[out] res The result of the vector matrix product (should not overlap with mult)
extern "C" __global__ void spRMult(
    const int rows,
    const int* rowStart,
    const int* columnIndex,
    const float* values,
    const float* mult,
    float* res
) {
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= rows) return;
    const int currentRowStart = rowStart[row];
    const int currentRowEnd = rowStart[row + 1];
    float sum = 0.0f;
    for(int i = currentRowStart; i < currentRowEnd; ++i) {
        const int column = columnIndex[i];
        sum += values[i] * mult[column];
    }
    res[row] = sum;
}


/// Multuply a matrix in CSR format with a dense vector and subtract this from a vector. Performing lhs - A * mult
/// @param[in] rows The number of rows of the matrix
/// @param[in] rowStart Array with length the number of rows + 1,
/// holding where each row starts in columnIndex and values arrays
/// @param[in] columnIndex elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]]
/// are the columns of the elements of the i-th row
/// @param[in] values elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]] are 
/// the values of the elements in the row.
/// @param[in] lhs The vector from which A * rhs will be subtracted (can overlap with res)
/// @param[in] mult The vector which multiples the matrix (should not overlap with res)
/// @param[out] res The result of the vector matrix product (should not overlap with rhs, can overlap with lhs)
extern "C" __global__ void spRMultSub(
    const int rows,
    const int* rowStart,
    const int* columnIndex,
    const float* values,
    const float* lhs,
    const float* mult,
    float* res
) {
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= rows) return;
    const int currentRowStart = rowStart[row];
    const int currentRowEnd = rowStart[row + 1];
    float sum = 0.0f;
    for(int i = currentRowStart; i < currentRowEnd; ++i) {
        const int column = columnIndex[i];
        sum += values[i] * mult[column];
    }
    res[row] = lhs[row] - sum;
}

/// Perform a * x + y where a is scalar, x and y are vectors. The result is stored in y
/// @param[in] vectorLength The number of elemens in both x and y vectors
/// @param[in] a The scalar which will multiply each element of x vector
/// @param[in] x x vector from the equation y = a * x + y
/// @param[inout] y y vector from the equation y = a * x + y. The result is stored in this vector
extern "C" __global__ void saxpy(
    const int vectorLength,
    const float a,
    const float* x,
    float* y
) {
  const unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < vectorLength) {
      y[i] = a*x[i] + y[i];
  }
}

/// Perform dot product between a and b vectors and store in result
/// @param[in] vectorLength The length of both a and b vectors
/// @param[in] a The first vector to dot
/// @param[in] b The second vector to dot
/// @param[out] result The result from dot(a, b)
extern "C" __global__ void dotProduct(
    const int vectorLength,
    const float* a,
    const float* b,
    float* result
) {
    __shared__ float cache[512];

    const unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
    const int cacheIndex = threadIdx.x;
    if(i < vectorLength) {
        cache[cacheIndex] = a[i] * b[i];
    } else {
        cache[cacheIndex] = 0.0f;
    }
    __syncthreads();

    for(int i = blockDim.x / 2; i > 0; i /= 2) {
        if(cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }
    if(cacheIndex == 0) {
        atomicAdd(result, cache[0]);
    }
}