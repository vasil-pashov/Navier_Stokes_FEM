#include "hip/hip_runtime.h"
#include "matrix_math_common.cuh"
#define HAS_COOP_GROUPS (__CUDA_ARCH__ >= 600)
#if HAS_COOP_GROUPS
    #include <hip/hip_cooperative_groups.h>
#endif

/// Multuply a matrix in CSR format with a dense vector. The vector is on the right hand side of the matrix.
/// @param[in] rows The number of rows of the matrix
/// @param[in] rowStart Array with length the number of rows + 1,
/// holding where each row starts in columnIndex and values arrays
/// @param[in] columnIndex elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]]
/// are the columns of the elements of the i-th row
/// @param[in] values elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]] are 
/// the values of the elements in the row.
/// @param[in] mult The vector which multiples the matrix (should not overlap with res)
/// @param[out] res The result of the vector matrix product (should not overlap with mult)
__device__ void spRMult(
    const int rows,
    const int* rowStart,
    const int* columnIndex,
    const float* values,
    const float* mult,
    float* res
) {
    unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    while(row < rows) {
        const int currentRowStart = rowStart[row];
        const int currentRowEnd = rowStart[row + 1];
        float sum = 0.0f;
        for(int i = currentRowStart; i < currentRowEnd; ++i) {
            const int column = columnIndex[i];
            sum += values[i] * mult[column];
        }
        res[row] = sum;
        row += gridDim.x * blockDim.x;
    }
}

extern "C" __global__ void spRMultKernel(
    const int rows,
    const int* rowStart,
    const int* columnIndex,
    const float* values,
    const float* mult,
    float* res
) {
    spRMult(rows, rowStart, columnIndex, values, mult, res);
}

/// Multuply a matrix in CSR format with a dense vector and subtract this from a vector. Performing lhs - A * mult
/// @param[in] rows The number of rows of the matrix
/// @param[in] rowStart Array with length the number of rows + 1,
/// holding where each row starts in columnIndex and values arrays
/// @param[in] columnIndex elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]]
/// are the columns of the elements of the i-th row
/// @param[in] values elements in range [columnIndex[rowStart[i]]]...columnIndex[rowStart[i+1]] are 
/// the values of the elements in the row.
/// @param[in] lhs The vector from which A * rhs will be subtracted (can overlap with res)
/// @param[in] mult The vector which multiples the matrix (should not overlap with res)
/// @param[out] res The result of the vector matrix product (should not overlap with rhs, can overlap with lhs)
__device__ void spRMultSub(
    const int rows,
    const int* rowStart,
    const int* columnIndex,
    const float* values,
    const float* lhs,
    const float* mult,
    float* res
) {
    unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    while(row < rows) {
        const int currentRowStart = rowStart[row];
        const int currentRowEnd = rowStart[row + 1];
        float sum = 0.0f;
        for(int i = currentRowStart; i < currentRowEnd; ++i) {
            const int column = columnIndex[i];
            sum += values[i] * mult[column];
        }
        res[row] = lhs[row] - sum;
        row += gridDim.x * blockDim.x;
    }
}

extern "C" __global__ void spRMultSubKernel(
    const int rows,
    const int* rowStart,
    const int* columnIndex,
    const float* values,
    const float* lhs,
    const float* mult,
    float* res
) {
    spRMultSub(rows, rowStart, columnIndex, values, lhs, mult, res);
}

/// Perform a * x + y where a is scalar, x and y are vectors. The result is stored in y
/// @param[in] vectorLength The number of elemens in both x and y vectors
/// @param[in] a The scalar which will multiply each element of x vector
/// @param[in] x x vector from the equation y = a * x + y
/// @param[inout] y y vector from the equation y = a * x + y. The result is stored in this vector
__device__ void saxpy(
    const int vectorLength,
    const float a,
    const float* x,
    float* y
) {
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  while(i < vectorLength) {
      y[i] += a*x[i];
      i += gridDim.x * blockDim.x;
  }
}

extern "C" __global__ void saxpyKernel(
    const int vectorLength,
    const float a,
    const float* x,
    float* y
) {
    saxpy(vectorLength, a, x, y);
}

/// Perform a * x + b * y where a and b are scalars and x and y are vectors.
/// @param[in] vectorLength The number of elements in vectors x and y
/// @param[in] a Scalar multiplier for the x vector
/// @param[in] b Scalar multiplier for the y vector
/// @param[in] x Vector multiplied by a
/// @param[in] y Vector multiplied by b
/// @param[out] result Vector where the result is stored
__device__ void saxpby(
    const int vectorLength,
    const float a,
    const float b,
    const float* x,
    const float* y,
    float* result
) {
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  while(i < vectorLength) {
      result[i] = a * x[i] + b * y[i];
      i += gridDim.x * blockDim.x;
  }
}
extern "C" __global__ void saxpbyKernel(
    const int vectorLength,
    const float a,
    const float b,
    const float* x,
    const float* y,
    float* result
) {
    saxpby(vectorLength, a, b, x, y, result);
}

/// Perform dot product between a and b vectors and store in result
/// @param[in] vectorLength The length of both a and b vectors
/// @param[in] a The first vector to dot
/// @param[in] b The second vector to dot
/// @param[out] result The result from dot(a, b)
__device__ void dotProduct(
    const int vectorLength,
    const float* a,
    const float* b,
    float* result
) {
    extern __shared__ float cache[];

    unsigned tid = blockIdx.x*blockDim.x + threadIdx.x;
    const int cacheIndex = threadIdx.x;
    float sum = 0.0f;
    while(tid < vectorLength) {
        sum += a[tid] * b[tid];
        tid += gridDim.x * blockDim.x;
    }
    cache[cacheIndex] = sum;
    __syncthreads();

    for(int i = blockDim.x / 2; i > 0; i >>= 1) {
        if(cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }
    if(cacheIndex == 0) {
        atomicAdd(result, cache[0]);
    }
}



extern "C" __global__ void dotProductKernel(
    const int vectorLength,
    const float* a,
    const float* b,
    float* result
) {
    dotProduct(vectorLength, a, b, result);
}

void __device__ syncGrid(unsigned int* barrier, unsigned int* generation) {
    if(threadIdx.x == 0) {
        volatile const unsigned int myGeneration = *generation;
        const unsigned int oldCount = atomicInc(barrier, gridDim.x - 1);
        if(oldCount == gridDim.x - 1) {
            atomicAdd(generation, 1);
        }
        while(atomicCAS(generation, myGeneration, myGeneration) == myGeneration);
    }
    __syncthreads();
}

extern "C" __global__ void conjugateGradientMegakernel(
    CGParams params
) {
    const int maxIterations = params.maxIterations;
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    const int rows = params.rows;
#if HAS_COOP_GROUPS
    using namespace cooperative_groups;
    grid_group grid = this_grid();
#endif
    for(int i = 0; i < maxIterations; ++i) {
        spRMult(rows, params.rowStart, params.columnIndex, params.values, params.p, params.ap);
        dotProduct(rows, params.ap, params.p, params.pAp);
#if HAS_COOP_GROUPS
        grid.sync();
#else
        syncGrid(params.barrier, params.generation);
#endif
        const float oldResidualNormSquared = *params.residualNormSquared;
        const float alpha = oldResidualNormSquared / *params.pAp;
        saxpy(rows, alpha, params.p, params.x);
        saxpy(rows, -alpha, params.ap, params.r);
        dotProduct(rows, params.r, params.r, params.newResidualNormSquared);
#if HAS_COOP_GROUPS
        grid.sync();
#else
        syncGrid(params.barrier, params.generation);
#endif
        const float newResidualNormSquared = *params.newResidualNormSquared;
        if(newResidualNormSquared < params.epsSq) {
            return;
        }
        const float beta = newResidualNormSquared / oldResidualNormSquared;
        saxpby(rows, 1, beta, params.r, params.p, params.p);
#if HAS_COOP_GROUPS
        grid.sync();
#else
        syncGrid(params.barrier, params.generation);
#endif
        if(tid == 0) {
            *params.residualNormSquared = newResidualNormSquared;
            *params.newResidualNormSquared = 0.0f;
            *params.pAp = 0.0f;
        }
#if HAS_COOP_GROUPS
        grid.sync();
#else
        syncGrid(params.barrier, params.generation);
#endif
    }
}